#include "hip/hip_runtime.h"
#include "cuda_bvh_builder.h"

// Return the number of p-sized blocks needed to cover N
template<typename T>
constexpr T NP2(T n, T p) {
    return ((n) + (p-1)) / p;
}


struct DeviceUniforms {
    int numTris;
    triangle* __restrict__ tris;

    // We use float4 rather than a float3 because you can load a float4 with one
    // instruction. If every thread in a warp loads a consecutive float4, all
    // 128 bytes can be loaded in a single memory transaction. So with
    // mins/indices in one buffer and maxs in another, we can load triangle data
    // across a warp with two total memory transactions.

    // [minX minY minZ (bitcast idx)]
    float4* __restrict__ triMinsIds;

    // [maxX maxY maxZ (undefined)]
    float4* __restrict__ triMaxs;
};

class CudaBVH {
public:
    CudaBVH(DeviceUniforms u)
      : _u(u) {}

    ~CudaBVH() {
        hipFree(_u.tris);
        hipFree(_u.triMinsIds);
        hipFree(_u.triMaxs);
    }
private:
    DeviceUniforms _u;
};

__global__ void setupTris(DeviceUniforms u) {
    int id = blockIdx.x*blockDim.x + threadIdx.x;
    if (id >= u.numTris) return;

    triangle& tri = u.tris[id];
    float3 p1 = {float(tri.p1.x), float(tri.p1.y), float(tri.p1.z)};
    float3 p2 = {float(tri.p2.x), float(tri.p2.y), float(tri.p2.z)};
    float3 p3 = {float(tri.p3.x), float(tri.p3.y), float(tri.p3.z)};

    float4 min = {
        fminf(p1.x, fminf(p2.x, p3.x)),
        fminf(p1.y, fminf(p2.y, p3.y)),
        fminf(p1.z, fminf(p2.z, p3.z)),
        __int_as_float(id)
    };
    u.triMinsIds[id] = min;

    float4 max {
        fmaxf(p1.x, fmaxf(p2.x, p3.x)),
        fmaxf(p1.y, fmaxf(p2.y, p3.y)),
        fmaxf(p1.z, fmaxf(p2.z, p3.z)),
        0.0f
    };
    u.triMaxs[id] = max;
}


std::shared_ptr<CudaBVH> build_cuda_bvh(
    triangle* tris, 
    int numTris
) {
    DeviceUniforms u;

    u.numTris = numTris;
    hipMalloc((triangle**)&u.tris, numTris*sizeof(triangle));
    hipMalloc((float4**)&u.triMinsIds, numTris*sizeof(float4));
    hipMalloc((float4**)&u.triMaxs, numTris*sizeof(float4));

    // Copy triangles into device memory
    hipMemcpy(
        u.tris,
        tris, 
        numTris*sizeof(triangle), 
        hipMemcpyHostToDevice);

    // Compute centroids, mins, and maxes for triangles
    setupTris<<<NP2(numTris, 64), 64>>>(u);

    // Synchronize
    hipDeviceSynchronize();

    return std::make_shared<CudaBVH>(u);
}